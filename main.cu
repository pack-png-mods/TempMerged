#include "hip/hip_runtime.h"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "hicpp-signed-bitwise"
//#define __JETBRAINS_IDE__
// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __constant__
#define __global__
#define __HIPCC__
#include <hip/device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif

#ifdef __INTELLISENSE__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#define __HIPCC__ //fixes function defenition in ide
//void __syncthreads();

#include <>
#include <hip/device_functions.h>
#include <device_atomic_functions.h>

#endif

#include <chrono>
#include <cstdint>
#include <thread>
#include <vector>
#include <atomic>
#include <iostream>
#include <iomanip>

#include "generator.h"

#define RANDOM_MULTIPLIER_LONG 0x5DEECE66DULL

#define Random uint64_t
#define RANDOM_MULTIPLIER RANDOM_MULTIPLIER_LONG
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK (1ULL << 48) - 1

// Random::next(bits)
__host__ __device__ inline uint32_t random_next(Random *random, int32_t bits) {
    *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
    return (uint32_t)(*random >> (48 - bits));
}

// Random::nextInt(bound)
__host__ __device__ inline uint32_t random_next_int(Random *random, uint32_t bound) {
    int32_t r = random_next(random, 31);
    int32_t m = bound - 1;
    if ((bound & m) == 0) {
        // Could probably use __mul64hi here
        r = (uint32_t)((bound * (uint64_t)r) >> 31);
    } else {
        r %= bound;
    }
    return r;
}

#define CHECK_GPU_ERR(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int32_t line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

// advance
#define advance_rng(rand, multiplier, addend) ((rand) = ((rand) * (multiplier) + (addend)) & RANDOM_MASK)
#define advance_16(rand) advance_rng(rand, 0x6DC260740241LL, 0xD0352014D90LL)
#define advance_m1(rand) advance_rng(rand, 0xDFE05BCB1365LL, 0x615C0E462AA9LL)
#define advance_m3759(rand) advance_rng(rand, 0x63A9985BE4ADLL, 0xA9AA8DA9BC9BLL)



#define WATERFALL_X 16
//#define WATERFALL_Y 76
#define WATERFALL_Z 10

#define TREE_X (WATERFALL_X - 5)
#define TREE_Z (WATERFALL_Z - 8)
#define TREE_HEIGHT 5

#define OTHER_TREE_COUNT 1
__device__ inline int32_t getTreeHeight(int32_t x, int32_t z) {
    if (x == TREE_X && z == TREE_Z)
        return TREE_HEIGHT;

    if (x == WATERFALL_X - 3 && z == WATERFALL_Z + 3)
        return 5;

    return 0;
}



#define MODULUS (1LL << 48)
#define X_TRANSLATE 0
#define L00 7847617LL
#define L01 (-18218081LL)
#define LI00 (24667315.0 / 16)
#define LI01 (18218081.0 / 16)
#define LI10 (-4824621.0 / 16)
#define LI11 (7847617.0 / 16)

#define CONST_MIN(a, b) ((a) < (b) ? (a) : (b))
#define CONST_MIN4(a, b, c, d) CONST_MIN(CONST_MIN(a, b), CONST_MIN(c, d))
#define CONST_MAX(a, b) ((a) > (b) ? (a) : (b))
#define CONST_MAX4(a, b, c, d) CONST_MAX(CONST_MAX(a, b), CONST_MAX(c, d))
#define CONST_FLOOR(x) ((x) < (int64_t) (x) ? (int64_t) (x) - 1 : (int64_t) (x))
#define CONST_CEIL(x) ((x) == (int64_t) (x) ? (int64_t) (x) : CONST_FLOOR((x) + 1))

// for a parallelogram ABCD https://media.discordapp.net/attachments/668607204009574411/671018577561649163/unknown.png
#define B_X LI00
#define B_Z LI10
#define C_X (LI00 + LI01)
#define C_Z (LI10 + LI11)
#define D_X LI01
#define D_Z LI11
#define LOWER_X CONST_MIN4(0, B_X, C_X, D_X)
#define LOWER_Z CONST_MIN4(0, B_Z, C_Z, D_Z)
#define UPPER_X CONST_MAX4(0, B_X, C_X, D_X)
#define UPPER_Z CONST_MAX4(0, B_Z, C_Z, D_Z)
#define ORIG_SIZE_X (UPPER_X - LOWER_X + 1)
#define SIZE_X CONST_CEIL(ORIG_SIZE_X - D_X)
#define SIZE_Z CONST_CEIL(UPPER_Z - LOWER_Z + 1)
#define TOTAL_WORK_SIZE (SIZE_X * SIZE_Z)

#define MAX_TREE_ATTEMPTS 12
#define MAX_TREE_SEARCH_BACK (3 * MAX_TREE_ATTEMPTS - 3 + 16 * OTHER_TREE_COUNT)

__constant__ uint64_t search_back_multipliers[MAX_TREE_SEARCH_BACK + 1];
__constant__ uint64_t search_back_addends[MAX_TREE_SEARCH_BACK + 1];
int32_t search_back_count;

#define WORK_UNIT_SIZE (1LL << 25)
#define BLOCK_SIZE 256

__global__ void doPreWork(uint64_t offset, Random* starts, int* num_starts) {
    // lattice tree position
    uint64_t global_id = blockIdx.x * blockDim.x + threadIdx.x;

    int64_t lattice_x = (int64_t) ((offset + global_id) % SIZE_X) + LOWER_X;
    int64_t lattice_z = (int64_t) ((offset + global_id) / SIZE_X) + LOWER_Z;
    lattice_z += (B_X * lattice_z < B_Z * lattice_x) * SIZE_Z;
    if (D_X * lattice_z > D_Z * lattice_x) {
        lattice_x += B_X;
        lattice_z += B_Z;
    }
    lattice_x += (int64_t) (TREE_X * LI00 + TREE_Z * LI01);
    lattice_z += (int64_t) (TREE_X * LI10 + TREE_Z * LI11);

    auto rand = (Random)((lattice_x * L00 + lattice_z * L01 + X_TRANSLATE) % MODULUS);
    advance_m1(rand);

    Random tree_start = rand;
    advance_m1(tree_start);

    bool res = random_next(&rand, 4) == TREE_X;
    res &= random_next(&rand, 4) == TREE_Z;
    res &= random_next_int(&rand, 3) == (uint64_t) (TREE_HEIGHT - 4);

    if (res) {
        int index = atomicAdd(num_starts, 1);
        starts[index] = tree_start;
    }
}

__global__ void doWork(const int32_t* num_starts, const Random* tree_starts, int32_t* num_seeds, uint64_t* seeds, int32_t gpu_search_back_count) {
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_starts; i += blockDim.x * gridDim.x) {
        Random tree_start = tree_starts[i];

        for (int32_t treeBackCalls = 0; treeBackCalls <= gpu_search_back_count; treeBackCalls++) {
            Random start = (tree_start * search_back_multipliers[treeBackCalls] + search_back_addends[treeBackCalls]) & RANDOM_MASK;
            Random rand = start;

            bool this_res = true;

            if (random_next_int(&rand, 10) == 0)
                continue;

            int32_t generated_tree[16];
            memset(generated_tree, 0x00, sizeof(generated_tree));

            int32_t treesMatched = 0;
            for (int32_t treeAttempt = 0; treeAttempt <= MAX_TREE_ATTEMPTS; treeAttempt++) {
                int32_t treeX = random_next(&rand, 4);
                int32_t treeZ = random_next(&rand, 4);
                int32_t wantedTreeHeight = getTreeHeight(treeX, treeZ);
                int32_t treeHeight = random_next_int(&rand, 3) + 4;

                int32_t& boolpack = generated_tree[treeX];
                const int32_t mask = 1 << (treeZ % 16);

                if (treeHeight == wantedTreeHeight && !(boolpack & mask)) {
                    treesMatched++;
                    boolpack |= mask;
                    advance_16(rand);
                }
            }

            this_res &= treesMatched >= OTHER_TREE_COUNT + 1;

            if (this_res) {
                Random start_chunk_rand = start;
                advance_m3759(start_chunk_rand);

                int32_t index = atomicAdd(num_seeds, 1);
                seeds[index] = start_chunk_rand;
            }

            advance_m1(start);
        }
    }
}

struct GPU_Node {
    int32_t* num_seeds;
    uint64_t* seeds;
    int32_t* num_tree_starts;
    Random* tree_starts;
};

void setup_gpu_node(GPU_Node* node, int32_t gpu) {
    CHECK_GPU_ERR(hipSetDevice(gpu));
    CHECK_GPU_ERR(hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds)));
    CHECK_GPU_ERR(hipMallocManaged(&node->seeds, (sizeof(Random)*WORK_UNIT_SIZE)));
    CHECK_GPU_ERR(hipMallocManaged(&node->num_tree_starts, sizeof(*node->num_tree_starts)));
    CHECK_GPU_ERR(hipMallocManaged(&node->tree_starts, (sizeof(Random)*WORK_UNIT_SIZE)));
}


#ifndef GPU_COUNT
#define GPU_COUNT 1
#endif

void calculate_search_backs() {
    bool allow_search_back[MAX_TREE_SEARCH_BACK + 1];
    memset(allow_search_back, false, sizeof(allow_search_back));

    for (int32_t i = 0; i <= MAX_TREE_ATTEMPTS - OTHER_TREE_COUNT - 1; i++) {
        allow_search_back[i * 3] = true;
    }

    for (int32_t tree = 0; tree < OTHER_TREE_COUNT; tree++) {
        for (int32_t i = 0; i <= MAX_TREE_SEARCH_BACK - 19; i++) {
            if (allow_search_back[i])
                allow_search_back[i + 19] = true;
        }
    }

    search_back_count = 0;
    uint64_t multiplier = 1;
    uint64_t addend = 0;
    uint64_t multipliers[MAX_TREE_SEARCH_BACK + 1];
    uint64_t addends[MAX_TREE_SEARCH_BACK + 1];
    for (int32_t i = 0; i <= MAX_TREE_SEARCH_BACK; i++) {
        if (allow_search_back[i]) {
            int32_t index = search_back_count++;
            multipliers[index] = multiplier;
            addends[index] = addend;
        }
        multiplier = (multiplier * 0xDFE05BCB1365LL) & RANDOM_MASK;
        addend = (0xDFE05BCB1365LL * addend + 0x615C0E462AA9LL) & RANDOM_MASK;
    }

    for (int32_t gpu = 0; gpu < GPU_COUNT; gpu++) {
        CHECK_GPU_ERR(hipSetDevice(gpu));
        CHECK_GPU_ERR(hipMemcpyToSymbol(HIP_SYMBOL(search_back_multipliers), &multipliers, search_back_count * sizeof(*multipliers)));
        CHECK_GPU_ERR(hipMemcpyToSymbol(HIP_SYMBOL(search_back_addends), &addends, search_back_count * sizeof(*addends)));
    }
}

#ifndef OFFSET
#define OFFSET 0
#endif

struct Thread {
    std::thread thread;
    std::vector<long long> threadBuffer;
};

int main(int argc, char *argv[]) {
    random_math::JavaRand::init();
    generator::ChunkGenerator::init();

    auto *nodes = (GPU_Node*)malloc(sizeof(GPU_Node) * GPU_COUNT);
    std::cout << "Searching " << TOTAL_WORK_SIZE << " total seeds...\n";

    calculate_search_backs();

    FILE* out_file = fopen("chunk_seeds.txt", "w");

    for (int32_t i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i], i);
    }

    std::vector<Thread> threads(std::thread::hardware_concurrency() - 4);

    std::atomic<uint64_t> count(0);
    auto lastIteration = std::chrono::system_clock::now();
    auto startTime = std::chrono::system_clock::now();
    long long* tempStorage = nullptr;
    uint64_t arraySize = 0;

    std::cout << "Using " << threads.size() << " threads for cpu work\n";

    for (uint64_t offset = OFFSET; offset < TOTAL_WORK_SIZE;) {

        for (int32_t gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));

            *nodes[gpu_index].num_tree_starts = 0;
            doPreWork<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(offset, nodes[gpu_index].tree_starts, nodes[gpu_index].num_tree_starts);
            offset += WORK_UNIT_SIZE;
        }

        for (int32_t gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));
            CHECK_GPU_ERR(hipDeviceSynchronize());
        }

        for (int32_t gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));

            *nodes[gpu_index].num_seeds = 0;
            doWork<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(nodes[gpu_index].num_tree_starts, nodes[gpu_index].tree_starts, nodes[gpu_index].num_seeds, nodes[gpu_index].seeds, search_back_count);
        }

        static auto threadFunc = [&](Thread &myThread, size_t start, size_t end) {
            for (int32_t j = start; j < end; ++j) {
                if (generator::ChunkGenerator::populate(tempStorage[j], X_TRANSLATE + 16)) {
                    myThread.threadBuffer.push_back(tempStorage[j]);
                    count++;
                }
            }
        };


        int32_t chunkSize = arraySize / threads.size();
        for(size_t i = 0; i < threads.size(); i++)
            threads[i].thread = std::thread(threadFunc, std::ref(threads[i]), i * chunkSize, (i == (threads.size() - 1)) ? arraySize : ((i + 1) * chunkSize));

        for(Thread& x : threads) {
            x.thread.join();

            for(const long long &val: x.threadBuffer)
                fprintf(out_file, "%lld\n", val);
            x.threadBuffer.clear();
        }

        fflush(out_file);
        free(tempStorage);

        tempStorage = (long long*)malloc(sizeof(long long));
        arraySize = 0;
        for (int32_t gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            CHECK_GPU_ERR(hipSetDevice(gpu_index));
            CHECK_GPU_ERR(hipDeviceSynchronize());
            tempStorage = (long long*) realloc(tempStorage, (*nodes[gpu_index].num_seeds + arraySize) * sizeof(long long));
            for (int32_t i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
                tempStorage[arraySize+i]=nodes[gpu_index].seeds[i];
            }
            arraySize += *nodes[gpu_index].num_seeds;
        }

        auto iterFinish = std::chrono::system_clock::now();
        std::chrono::duration<double> iterationTime = iterFinish - lastIteration;
        std::chrono::duration<double> elapsedTime = iterFinish - startTime;
        lastIteration = iterFinish;
        uint64_t numSearched = offset + WORK_UNIT_SIZE * GPU_COUNT - OFFSET;
        double speed = numSearched / elapsedTime.count() / 1000000;
        double progress = (double)numSearched / (double)TOTAL_WORK_SIZE * 100.0;
        double estimatedTime = (double)(TOTAL_WORK_SIZE - numSearched) / speed / 1000000;
        uint64_t curCount = count;
        char suffix = 's';
        if (estimatedTime >= 3600) {
            suffix = 'h';
            estimatedTime /= 3600.0;
        } else if (estimatedTime >= 60) {
            suffix = 'm';
            estimatedTime /= 60.0;
        }
        if (progress >= 100.0) {
            estimatedTime = 0.0;
            suffix = 's';
        }
        std::cout << "Searched: " << std::setw(13) << numSearched << " seeds. Found: " << std::setw(13) << count.load() << " matches. Uptime: " <<
                std::fixed << std::setprecision(1) << elapsedTime.count() << "s. Speed: " << std::fixed <<
                std::setprecision(2) << speed << "m seeds/s. Completion: " << std::setprecision(2) << progress <<
                "%. ETA: " << std::fixed << std::setprecision(2) << estimatedTime << suffix << ".\n";
    }

    // Last batch to do
    for (int32_t j = 0; j < arraySize; ++j) {
        if (generator::ChunkGenerator::populate(tempStorage[j], X_TRANSLATE + 16)) {
            fprintf(out_file, "%lld\n", tempStorage[j]);
            count++;
        }
    }

    fflush(out_file);
    free(tempStorage);
    fclose(out_file);
}
